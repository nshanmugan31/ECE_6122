/*
Author: Nishalini Shanmugan
Class: ECE4122 
Last Date Modified: 11/21/2021

Description:

In this lab, we used a C\C++ program using CUDA to solve the steady state temperature distribution in the thin plate. 
We are passing in the number of interior points in the array and the number of iterations, and returning the number
of miliseconds it took to calculate the solution using CUDA events. We are also writing the final temperature values
into an finalTemperature.csv file. 

*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <list>
#include <sstream>
#include <fstream>
#include <stdio.h>
#include <string>
#include <ctype.h>
#include <stdio.h>
#include <iomanip>


using namespace std;

//Call the kernal and fill the data with Jacobi Iteration
__global__ void distributionPlate(double*h, double*g, unsigned long n) 
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if((i > 0) && (i < n-1) && (j > 0) && (j < n-1))
    {
        g[i*n+j] = 0.25*(h[(i-1)*n + j] + h[(i+1)*n + j]+h[i*n + (j-1)] + h[i*n +(j+1)]);
    }

}

//Create the array
void populateArray(double *a, int dim)
{
    for(int i=0;i<dim;i++)
    {
        for(int j=0; j<dim;j++)
        {
            //If the first row is greater than 30% or less than 70%, add 100 into each element
            if ((j > 0.3*(dim-1)) && (j < 0.7*(dim-1)) && (i==0))
            {
                a[i*dim+j]=100;
            }
            //Otherwise add 20 to all the elements
            else
            {
                a[i*dim+j]=20;
            }
        }
    }
}

//Print the array to the finalTemperature.csv file
void printArray(double* a, int dim)
{
    std::ofstream myfile;
    //Open a file called server.log and append to it using ios::app
    myfile.open("finalTemperature.csv", std::ios::out | std::ios::trunc);
    string row;
    for(int i=0; i < dim; i++)
    {
        for(int j=0; j < dim; j++)
        {
            row +=to_string(a[i*dim+j]);
            //enter if the end of the row is reached
            if(j == dim-1)
            {
                row +="\n";
            }
            //Otherwise put a comma in between all the comments.
            else
            {
                row +=",";
            }
        }
    }
    myfile << row << std::endl;
    myfile.close();
}

int main(int argc, char* argv[])
{
    int n = 0;
    int limit = 0;
    
    //If there are 5 arguments continue, else return invalid command arguments
    if (argc == 5)
    {   
        if ((atoi(argv[2])) || (atoi(argv[4])))
        {
            n = std::stoi(argv[2]);
            limit = std::stoi(argv[4]);
            //If the number of N x N interior points and the number of iterations is less than 0, return invalid parameter
            if (n < 0 || limit < 0)
            {
                std::cout << "Invalid parameter, please check your values." <<std::endl;
            }
            else
            {
            //(n+2)*(n+2)
            const int dim = (n+2);
            const int length = dim*dim;
            const int size = sizeof(double)*(length);
            
            //allocate memory
            double*h = (double*) malloc(size);
            double*g = (double*) malloc(size); 

            //Declare and cuda host copies
            double *hd;
            double *gd;
            hipMalloc((void**)&hd, size);
            hipMalloc((void**)&gd, size);
            
            //fill host array with data
            populateArray(h,dim);
            populateArray(g,dim);
            
            //GPU timing
            float gpu_elapsed_time;
            hipEvent_t gpu_start, gpu_stop;
            hipEventCreate(&gpu_start);
            hipEventCreate(&gpu_stop);

            //copy from host to device
            hipEventRecord(gpu_start, 0);
            hipMemcpy(hd, h, size, hipMemcpyHostToDevice);
            hipMemcpy(gd, g, size, hipMemcpyHostToDevice);

            //call kernel
            dim3 blockSize(16,16);
            dim3 gridSize(n/16+1, n/16+1);
            
            for (int iteration = 0; iteration < limit; iteration++)
            {
                // Launch distributioPlate() kernel on GPU with gridSize blocks 
                distributionPlate<<< gridSize, blockSize >>>(hd, gd, dim);
                
                //Use swap function to swap the arrays
                swap(hd,gd);
            }

            //copy from device to host
            hipMemcpy(h, hd, size, hipMemcpyDeviceToHost);

            //Track the gpu start, stop, and elapsed time
            hipEventRecord(gpu_stop,0);
            hipEventSynchronize(gpu_stop);
            hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
            hipEventDestroy(gpu_start);
            hipEventDestroy(gpu_stop);
            
            //Output to the console the number of milliseconds it took to calculate the solution using CUDA events
            std::cout << std::fixed;
            std::cout << "Number of Miliseconds using CUDA events:" << std::setprecision(2) << gpu_elapsed_time <<std::endl;
            printArray(h,dim);

            //free memory
            free(h);
            free(g);
            hipFree(hd);
            hipFree(gd);
            return 0;   
            }
                
        }
        //If they are unable to convert, return invalid parameter
        else
        {
            std::cout << "Invalid parameter, please check your values." <<std::endl;
        }
        
    }
    //Invalid command line arguments entered - Anything except 5 arguments. 
    else
    {
        std::cout << "Invalid command line arguments!" << std::endl;
    }
    return 0;
}
